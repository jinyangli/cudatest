#include "hip/hip_runtime.h"
#include "util/cuPrintf.cu"

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <assert.h>


#include "mem.h"
#include "timer.h"

extern int debug;


__global__  void kernel(float *array, int n, int stride)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x; 

	int start = index * stride;
	int end = (index+1) * stride;
	if (end > n)
		end = n;

	for (int i = start; i < end; i++) {
		array[i] = sqrtf(array[i]);
	}

	cuPrintf("n %d stride %d my_id %d start %d end %d array[0]=%f\n", n, stride, index, start, end, array[0]);
}

void launch_kernel(int n_tblk, int nt_tblk, float *device, int n)
{	

	if (debug) cudaPrintfInit(); // initialize cuPrintf

	{
		Timer t("Kernel finished ");

		kernel<<<n_tblk,nt_tblk>>>(device, n, n/(n_tblk*nt_tblk));
		hipDeviceSynchronize();
	}

	if (debug) {
		// display the device's greeting
		cudaPrintfDisplay();
			
		// clean up after cuPrintf
		cudaPrintfEnd();
	}

}

void alloc_mem(float **host_array, float **device_array, int n)
{
	hipError_t err = hipSetDeviceFlags(hipDeviceMapHost);
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	hipHostMalloc(host_array, n*sizeof(float));
	memset(*host_array, 0, n*sizeof(float));

	// hipMalloc device memory
	//hipMalloc(device_array, n* sizeof(float));
	assert(hipHostGetDevicePointer(device_array, *host_array, 0) == hipSuccess);

	// zero out the device array with hipMemset
	hipMemset(*device_array, 0, n* sizeof(float));

}

void transfer_mem(float *device, float *host, int n, bool host2dev)
{
	struct timespec t0, t1;

	clock_gettime(CLOCK_REALTIME, &t0);
	if (host2dev) 
		hipMemcpy(device, host, n* sizeof(float), hipMemcpyHostToDevice);
	else
		hipMemcpy(host, device, n* sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &t1);
	printf("%s Transfer took %ld usec\n", host2dev?"H->D":"D->H", TIME_DIFF(t0, t1));
}

void copy_mem(float *dst, float *src, int n)
{
	struct timespec t0, t1;

	clock_gettime(CLOCK_REALTIME, &t0);
	hipMemcpy(dst, src, n* sizeof(float), hipMemcpyDefault);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &t1);

	struct hipPointerAttribute_t attr;
	assert(hipPointerGetAttributes(&attr, dst)==hipSuccess);
	printf("%s Transfer took %ld usec\n", (attr.memoryType == hipMemoryTypeHost)?"H->D":"D->H", TIME_DIFF(t0, t1));
}

void free_mem(float *host, float *device)
{
	hipHostFree(host);
	hipFree(device);
}


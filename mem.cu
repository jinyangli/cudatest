#include "hip/hip_runtime.h"
#include "util/cuPrintf.cu"

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <mpi.h>

__global__  void kernel(float *array, int n, int stride)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x; 

	int start = index * stride;
	int end = (index+1) * stride;
	if (end > n)
		end = n;

	for (int i = start; i < end; i++) {
		array[i] = sqrtf(array[i]);
	}

	cuPrintf("n %d stride %d my_id %d start %d end %d array[0]=%f\n", n, stride, index, start, end, array[0]);
}

int
timediff(struct timespec *t0, struct timespec *t1)
{
	return (t1->tv_sec - t0->tv_sec)*1000000 + (t1->tv_nsec - t0->tv_nsec)/1000;
}

int main(int argc, char **argv)
{
	bool debug = 1;
	int num_elements = 16;
	int n_tblk = 1;
	int nt_tblk = 1;
	char c;
	struct timespec t0, t1;

	while (( c = getopt(argc, argv, "d:n:b:t:")) != -1) {
		switch (c) 
		{
			case 'd':
				debug = atoi(optarg);
				break;
			case 'n':
				if (num_elements > 250) {
					num_elements = 250;
					printf("Capped to 250M elements!!!!\n");
				}
				num_elements = atoi(optarg) * 1000000;
				break;
			case 'b':
				n_tblk= atoi(optarg);
				break;
			case 't':
				nt_tblk= atoi(optarg);
				break;
			default:
				printf("valid options: -n <size> -b <# thread blocks> -t <# threads per block> -d <debug>\n");
				exit(1);
			}
	}

	printf("Array size:%dM, ThreadBlocks:%d, ThreadsPerBlock: %d Total Threads %d\n", \
			num_elements/1000000, n_tblk, nt_tblk, n_tblk * nt_tblk);

	int myrank;
	MPI_Status status;
	int tag = 999;

	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &myrank);

	float *device_array = 0;
	float *host_array = 0;

	// malloc host memory
	host_array = (float *)malloc(num_elements * sizeof(float));
	for (int i = 0; i < num_elements; i++) {
		host_array[i] = i + 4.0;
	}

	// hipMalloc device memory
	hipMalloc((void**)&device_array, num_elements * sizeof(float));

	// zero out the device array with hipMemset
	hipMemset(device_array, 0, num_elements * sizeof(float));

	if (myrank == 0 ) {
		// copy the contents of the device array to the host
		clock_gettime(CLOCK_REALTIME, &t0);
		hipMemcpy(device_array, host_array, num_elements * sizeof(float), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		clock_gettime(CLOCK_REALTIME, &t1);
		printf("H->D Transfer %d usec\n", timediff(&t0, &t1));

		if (debug) {
			cudaPrintfInit(); // initialize cuPrintf
		}


		clock_gettime(CLOCK_REALTIME, &t0);
		kernel<<<n_tblk,nt_tblk>>>(device_array, num_elements, num_elements/(n_tblk*nt_tblk));
		hipDeviceSynchronize();
		clock_gettime(CLOCK_REALTIME, &t1);

		printf("Kernel finished in %d usec %f\n", timediff(&t0, &t1), host_array[0]);

		clock_gettime(CLOCK_REALTIME, &t0);
		hipMemcpy(host_array, device_array, num_elements * sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		clock_gettime(CLOCK_REALTIME, &t1);
		printf("D->H Transfer %d usec\n", timediff(&t0, &t1));

		MPI_Send(device_array, num_elements, MPI_FLOAT, 1, tag, MPI_COMM_WORLD);
		printf("MPI sent\n");
	}else {
		MPI_Recv(host_array, num_elements, MPI_FLOAT, 0, tag, MPI_COMM_WORLD, &status);
		int count;
		MPI_Get_count(&status, MPI_FLOAT, &count); 
		printf("MPI received %d floats %f\n", count, host_array[0]);
	}

	if (debug) {
		// display the device's greeting
		cudaPrintfDisplay();
			
		// clean up after cuPrintf
		cudaPrintfEnd();
	}

	if (debug) {
		for(int i = 0; i < (num_elements<10?num_elements:10); ++i)
			printf("%f ", host_array[i]);
		printf("\n");
	}

	// use free to deallocate the host array
	free(host_array);

	// use hipFree to deallocate the device array
	hipFree(device_array);

	MPI_Finalize();

	return 0;
}
